#include "depth_utils.h"
#include "ros/ros.h"
#include "marvin_convnet/DetectObjects.h"
#include "realsense_camera/StreamSensor.h"
#include <opencv2/opencv.hpp>

// Marvin
#define DATATYPE 0
#include "marvin.hpp"

std::string shelf_net_arch_filename = "/home/andyz/apc/toolbox/ros-packages/catkin_ws/src/marvin_convnet/net/shelf_tote_testing.json";
std::string tote_net_arch_filename = "/home/andyz/apc/toolbox/ros-packages/catkin_ws/src/marvin_convnet/net/shelf_tote_testing.json";
std::string shelf_net_weights_filename = "/home/andyz/apc/toolbox/ros-packages/catkin_ws/src/marvin_convnet/net/shelf_net.marvin";
std::string tote_net_weights_filename = "/home/andyz/apc/toolbox/ros-packages/catkin_ws/src/marvin_convnet/net/tote_net.marvin";

// Service modes and names
const int READ_SENSOR_NO_FORWARD_PASS = 0;
const int READ_SENSOR_FORWARD_PASS = 1;
const int READ_FILE_FORWARD_PASS = 2;
int service_mode;
std::string service_name;
std::string camera_service_name;

// Directory to read/write all RGB-D files and response maps
std::string read_directory;
std::string write_directory;
std::string net_directory;

// Global buffers for sensor data retrieval
int frame_width = 640;
int frame_height = 480;
bool cloud_buffer_loaded = false;
float * cloud_buffer_pts = new float[frame_width * frame_height * 3];
uint8_t * cloud_buffer_rgb = new uint8_t[frame_width * frame_height * 3];
float * buffer_depth = new float[frame_width * frame_height];
float * buffer_raw_depth = new float[frame_width * frame_height];
float * color_cam_intrinsics = new float[9];
float * depth_cam_intrinsics = new float[9];
float * depth2color_extrinsics = new float[12];

// Load Marvin FCN network architectures
marvin::Net shelf_net(shelf_net_arch_filename);
marvin::Net tote_net(tote_net_arch_filename);

// Marvin responses
StorageT* color_data_CPU = NULL;
StorageT* prob_CPU_StorageT = NULL;
ComputeT* prob_CPU_ComputeT = NULL;

ros::ServiceClient client_sensor;

const int num_apc_objects = 39;

std::string shelf_bin_ids = "ABCDEFGHIJKL";

// Service call
bool srv_detect(marvin_convnet::DetectObjects::Request  &req,
                marvin_convnet::DetectObjects::Response &res) {
  ROS_INFO("Recieved service request.");

  int bin_id = req.BinId;
  int frame_id = req.FrameId;
  res.FrameId = frame_id;

  // Get frame filenames
  std::ostringstream frame_prefix;
  frame_prefix << std::setw(6) << std::setfill('0') << frame_id;
  std::string color_frame_filename = "/frame-" + frame_prefix.str() + ".color.png";
  std::string depth_frame_filename = "/frame-" + frame_prefix.str() + ".depth.png";
  std::string raw_depth_frame_filename = "/raw/frame-" + frame_prefix.str() + ".depth.png";

  // Retrieve RGB-D data from camera service and save to disk
  if (service_mode == READ_SENSOR_FORWARD_PASS || service_mode == READ_SENSOR_NO_FORWARD_PASS) {
    realsense_camera::StreamSensor srv_sensor;
    if (!client_sensor.call(srv_sensor)) {
      std::cout << "Failed to call service " + camera_service_name << std::endl;
      return true;
    }

    // Load point cloud and depth buffers from camera service
    std::copy(srv_sensor.response.cloudXYZ.begin(), srv_sensor.response.cloudXYZ.end(), cloud_buffer_pts);
    std::copy(srv_sensor.response.cloudRGB.begin(), srv_sensor.response.cloudRGB.end(), cloud_buffer_rgb);
    std::copy(srv_sensor.response.rawDepth.begin(), srv_sensor.response.rawDepth.end(), buffer_raw_depth);
    for (int i = 0; i < frame_width * frame_height * 3; i += 3)
      buffer_depth[i / 3] = (float)(srv_sensor.response.cloudXYZ[i + 2]);
    // cloud_buffer_loaded = true;

    // Load camera information from camera service
    for (int i = 0; i < 9; ++i) {
      color_cam_intrinsics[i] = srv_sensor.response.colorCamIntrinsics[i];
      depth_cam_intrinsics[i] = srv_sensor.response.depthCamIntrinsics[i];
    }
    for (int i = 0; i < 12; ++i)
      depth2color_extrinsics[i] = srv_sensor.response.depth2colorExtrinsics[i];

    // Save Bin ID
    std::string cam_info_file = write_directory + "/cam.info.txt";
    FILE *fp = fopen(cam_info_file.c_str(), "w");
    if (bin_id == -1)
      fprintf(fp, "# Environment: tote\n# Bin ID: N/A\n");
    else
      fprintf(fp, "# Environment: shelf\n# Bin ID: %s\n", shelf_bin_ids.substr(bin_id, 1).c_str());

    // Save object list
    fprintf(fp, "# Objects: [");
    for (int i = 0; i < (req.ObjectNames.size() - 1); ++i)
      fprintf(fp, "\"%s\",", req.ObjectNames[i].c_str());
    if (req.ObjectNames.size() > 0)
      fprintf(fp, "\"%s\"",req.ObjectNames[req.ObjectNames.size() - 1].c_str());
    fprintf(fp, "]\n\n");

    // Save camera intrinsics of color sensor
    fprintf(fp, "# Color camera intrinsic matrix\n");
    for (int i = 0; i < 3; ++i)
      fprintf(fp, "%15.8e\t %15.8e\t %15.8e\t\n", (float)(color_cam_intrinsics[i * 3 + 0]), (float)(color_cam_intrinsics[i * 3 + 1]), (float)(color_cam_intrinsics[i * 3 + 2]));

    // Save camera intrinsics of depth sensor
    fprintf(fp, "\n# Depth camera intrinsic matrix\n");
    for (int i = 0; i < 3; ++i)
      fprintf(fp, "%15.8e\t %15.8e\t %15.8e\t\n", (float)(depth_cam_intrinsics[i * 3 + 0]), (float)(depth_cam_intrinsics[i * 3 + 1]), (float)(depth_cam_intrinsics[i * 3 + 2]));

    // Save camera-to-camera extrinsics from depth sensor to color sensor
    fprintf(fp, "\n# Depth-to-color camera extrinsic matrix\n");
    for (int i = 0; i < 3; ++i)
      fprintf(fp, "%15.8e\t %15.8e\t %15.8e\t %15.8e\t\n", (float)(depth2color_extrinsics[i * 4 + 0]), (float)(depth2color_extrinsics[i * 4 + 1]), (float)(depth2color_extrinsics[i * 4 + 2]), (float)(depth2color_extrinsics[i * 4 + 3]));
    fprintf(fp, "%15.8e\t %15.8e\t %15.8e\t %15.8e\t\n", 0.0f, 0.0f, 0.0f, 1.0f);
    fclose(fp);

    // Save color frame
    cv::Mat color_frame = cv::Mat(frame_width * frame_height * 3, 1, CV_8U, cloud_buffer_rgb).clone();
    color_frame = color_frame.reshape(3, frame_height);
    cv::cvtColor(color_frame, color_frame, CV_RGB2BGR);
    cv::imwrite(write_directory + color_frame_filename, color_frame);
    cloud_buffer_rgb = color_frame.data;

    // Save depth frame (aligned and un-aligned raw)
    WriteDepth(write_directory + depth_frame_filename, buffer_depth, frame_height, frame_width); 
    WriteDepth(write_directory + raw_depth_frame_filename, buffer_raw_depth, frame_height, frame_width);
  } else {

    // Read color frame
    cv::Mat color_frame = cv::imread(read_directory + color_frame_filename, CV_LOAD_IMAGE_COLOR);
    cloud_buffer_rgb = color_frame.data;
  }

  if (service_mode == READ_SENSOR_FORWARD_PASS || service_mode == READ_FILE_FORWARD_PASS) {

    // Color: BGR format, mean subtracted
    for (int r = 0; r < frame_height; ++r)
      for (int c = 0; c < frame_width; ++c) {
        color_data_CPU[0 * frame_height * frame_width + r * frame_width + c] = CPUCompute2StorageT(ComputeT(cloud_buffer_rgb[0 + 3 * (c + frame_width * r)]) - ComputeT(102.9801f)); // B
        color_data_CPU[1 * frame_height * frame_width + r * frame_width + c] = CPUCompute2StorageT(ComputeT(cloud_buffer_rgb[1 + 3 * (c + frame_width * r)]) - ComputeT(115.9465f)); // G
        color_data_CPU[2 * frame_height * frame_width + r * frame_width + c] = CPUCompute2StorageT(ComputeT(cloud_buffer_rgb[2 + 3 * (c + frame_width * r)]) - ComputeT(122.7717f)); // R
        // color_data_CPU[0 * frame_height * frame_width + r * frame_width + c] = CPUCompute2StorageT(ComputeT(cloud_buffer_rgb[0 * frame_height * frame_width + r * frame_width + c]) - ComputeT(102.9801f)); // B
        // color_data_CPU[1 * frame_height * frame_width + r * frame_width + c] = CPUCompute2StorageT(ComputeT(cloud_buffer_rgb[1 * frame_height * frame_width + r * frame_width + c]) - ComputeT(115.9465f)); // G
        // color_data_CPU[2 * frame_height * frame_width + r * frame_width + c] = CPUCompute2StorageT(ComputeT(cloud_buffer_rgb[2 * frame_height * frame_width + r * frame_width + c]) - ComputeT(122.7717f)); // R
        // ROS_INFO("%f",CPUStorage2ComputeT(color_data_CPU[0 * frame_height * frame_width + r * frame_width + c]));
        // ROS_INFO("%f",CPUStorage2ComputeT(color_data_CPU[1 * frame_height * frame_width + r * frame_width + c]));
        // ROS_INFO("%f",CPUStorage2ComputeT(color_data_CPU[2 * frame_height * frame_width + r * frame_width + c]));
      } 

    // for (int r = 0; r < frame_height; ++r)
    //   for (int c = 0; c < frame_width; ++c) {
    //     rgbCPUStorageT[ 0 * frame_height * frame_width + r * frame_width + c] = CPUCompute2StorageT(ComputeT( rgb_uint8[ 0 + 3 * (c + frame_width * r) ] ) - ComputeT(102.9801f)); // B
    //     rgbCPUStorageT[ 1 * frame_height * frame_width + r * frame_width + c] = CPUCompute2StorageT(ComputeT( rgb_uint8[ 1 + 3 * (c + frame_width * r) ] ) - ComputeT(115.9465f)); // G
    //     rgbCPUStorageT[ 2 * frame_height * frame_width + r * frame_width + c] = CPUCompute2StorageT(ComputeT( rgb_uint8[ 2 + 3 * (c + frame_width * r) ] ) - ComputeT(122.7717f)); // R
    //   }

    // Run forward pass through marvin FCN
    ROS_INFO("Forward Marvin to get segmentation results.");
    marvin::Response * rData;
    marvin::Response * rProb;
    if (bin_id == -1) {
      rData = tote_net.getResponse("data");
      rProb = tote_net.getResponse("prob");
    } else {
      rData = shelf_net.getResponse("data");
      rProb = shelf_net.getResponse("prob");
    }
    hipMemcpy(rData->dataGPU, color_data_CPU, rData->numBytes(), hipMemcpyHostToDevice);
    if (bin_id == -1)
      tote_net.forward();
    else
      shelf_net.forward();
    hipMemcpy(prob_CPU_StorageT, rProb->dataGPU, rProb->numBytes(), hipMemcpyDeviceToHost);
    for (int i = 0; i < frame_height * frame_width * (num_apc_objects + 1); ++i)
      prob_CPU_ComputeT[i] = CPUStorage2ComputeT(prob_CPU_StorageT[i]);

    // Get full object list
    std::vector<std::string> all_object_names = {"background", "barkely_hide_bones", "cherokee_easy_tee_shirt", "clorox_utility_brush", "cloud_b_plush_bear", "cool_shot_glue_sticks", "command_hooks", "crayola_24_ct", "creativity_chenille_stems", "dasani_water_bottle",
                                                 "dove_beauty_bar", "dr_browns_bottle_brush", "easter_turtle_sippy_cup", "elmers_washable_no_run_school_glue", "expo_dry_erase_board_eraser", "fiskars_scissors_red", "fitness_gear_3lb_dumbbell", "folgers_classic_roast_coffee", "hanes_tube_socks", "i_am_a_bunny_book",
                                                 "jane_eyre_dvd", "kleenex_paper_towels", "kleenex_tissue_box", "kyjen_squeakin_eggs_plush_puppies", "laugh_out_loud_joke_book", "oral_b_toothbrush_green", "oral_b_toothbrush_red", "peva_shower_curtain_liner", "platinum_pets_dog_bowl", "rawlings_baseball",
                                                 "rolodex_jumbo_pencil_cup", "safety_first_outlet_plugs", "scotch_bubble_mailer", "scotch_duct_tape", "soft_white_lightbulb", "staples_index_cards", "ticonderoga_12_pencils", "up_glucose_bottle", "womens_knit_gloves", "woods_extension_cord"};
    std::vector<std::string> selected_object_names = req.ObjectNames;

    // Remove duplicates in selected object list
    std::sort(selected_object_names.begin(), selected_object_names.end());
    selected_object_names.erase(std::unique(selected_object_names.begin(), selected_object_names.end()), selected_object_names.end());

    // Loop through each object in selected list
    for (int selected_idx = 0; selected_idx < selected_object_names.size(); selected_idx++) {
      std::string curr_object_name = selected_object_names[selected_idx];
      int curr_object_idx = std::distance(all_object_names.begin(), find(all_object_names.begin(), all_object_names.end(), curr_object_name));

      std::vector<ComputeT> predMap_object(prob_CPU_ComputeT + curr_object_idx * frame_height * frame_width, prob_CPU_ComputeT + (curr_object_idx + 1) * frame_height * frame_width);

      // Write result to binary file
      std::string result_filename = write_directory + "/frame-" + frame_prefix.str() + "." + all_object_names[curr_object_idx] + ".mask.bin";
      std::ofstream out_file(result_filename, std::ios::binary | std::ios::out);
      for (int i = 0; i < frame_height * frame_width; i++)
        out_file.write((char*)&predMap_object[i], sizeof(float));
      out_file.close();
    }
  }
  return true;
}

int main(int argc, char **argv) {

  // Setup ROS
  ros::init(argc, argv, "marvin_convnet", ros::init_options::AnonymousName);
  ros::NodeHandle n;
  ros::NodeHandle priv_nh("~");

  // Get service parameters
  priv_nh.param("service_name", service_name, std::string("marvin_convnet"));
  priv_nh.param("service_mode", service_mode, READ_SENSOR_NO_FORWARD_PASS);
  priv_nh.param("camera_service_name", camera_service_name, std::string("/realsense_camera"));
  priv_nh.param("read_directory", read_directory, std::string(""));
  priv_nh.param("write_directory", write_directory, std::string(""));
  priv_nh.param("net_directory", net_directory, std::string(""));

  // Assert parameters
  assert (!write_directory.empty());
  if (service_mode == READ_FILE_FORWARD_PASS)
    assert (!read_directory.empty());

  // Create a data folder to save RGB-D frames
  std::ifstream file(write_directory);
  if (file.fail())
    system(std::string("mkdir -p " + write_directory + "/raw").c_str());

  // Start service
  ros::ServiceServer service_detect = n.advertiseService(service_name, srv_detect);
  ROS_INFO("Writing data to directory: %s", write_directory.c_str());

  // Connect to Realsense camera
  if (service_mode == READ_SENSOR_FORWARD_PASS || service_mode == READ_SENSOR_NO_FORWARD_PASS) {
    ROS_INFO("Reading data from camera service: %s", camera_service_name.c_str());
    client_sensor = n.serviceClient<realsense_camera::StreamSensor>(camera_service_name);
  } else
    ROS_INFO("Reading data from directory: %s", read_directory.c_str());

  // Setup Marvin
  if (service_mode == READ_SENSOR_FORWARD_PASS || service_mode == READ_FILE_FORWARD_PASS) {
    ROS_INFO("Loading Marvin.");
    shelf_net.Malloc(marvin::Testing);
    tote_net.Malloc(marvin::Testing);
    shelf_net.loadWeights(shelf_net_weights_filename);
    tote_net.loadWeights(tote_net_weights_filename);
    color_data_CPU = new StorageT[frame_width * frame_height * 3];
    prob_CPU_StorageT = new StorageT[frame_width * frame_height * (num_apc_objects + 1)];
    prob_CPU_ComputeT = new ComputeT[frame_height * frame_width * (num_apc_objects + 1)];
  }

  ROS_INFO("Ready.");
  ros::spin();

  return 0;
}

